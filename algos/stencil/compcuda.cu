#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <sys/time.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include <homm.h>
#include "cos.h"

#ifdef USE_CUDA

__global__ void gpu_kernel(REAL *afrom, REAL *ato, long nx, long ny, long lz0, long lz1)
{
  long ix = blockIdx.x*blockDim.x + threadIdx.x;
  long iy = blockIdx.y*blockDim.y + threadIdx.y;
  if (ix < 1 || ix >= nx+1) {
    return; // do nothing
  }

  if (iy < 1 || iy >= ny+1) {
    return; // do nothing
  }

  long iz;
  for (iz = lz0; iz < lz1; iz++) {
    // update one point
    KERNEL(ix, iy, iz, nx, ny, afrom, ato);
  }
  
  return;
}
    
int update(REAL *afrom, REAL *ato, vec3 v0, vec3 v1)
{
  dim3 bs = dim3(32, 32, 1);
  dim3 gs = dim3(((v1.x-v0.x+2)+(bs.x-1))/bs.x, ((v1.y-v0.y+2)+(bs.y-1))/bs.y);

  gpu_kernel<<<gs, bs>>>(afrom, ato, n3d.x, n3d.y, v0.z, v1.z);
#if 0
  hipError_t crc = hipDeviceSynchronize();
  if (crc != hipSuccess) {
    fprintf(stderr, "Error in update(): afrom=%p, ato=%p, z=[%ld,%ld)\n",
	    afrom, ato, v0.z, v1.z);
  }
  CERR(crc);
#endif
  return 0;
}

#endif // GPU
